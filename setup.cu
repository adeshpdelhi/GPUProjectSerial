#ifndef SETUP_H
#define SETUP_H

#include "render.h"
// constants
const unsigned int window_width  = 1366;
const unsigned int window_height = 768;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

///////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Collision Detection");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    // glDisable(GL_DEPTH_TEST);
    glEnable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    // gluPerspective(60.0, 10.0, 10.0, 20.0);

    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}


#endif