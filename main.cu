#include "hip/hip_runtime.h"
#include "render.cu"
// #include "object.h"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    host_pos = (float4 *)malloc(MAX_MAPPINGS*sizeof(float4));
    if(host_pos == NULL){
        printf("Error: Unable to allocate mapping memory on host.\n");
        exit(-1);
    }
    srand((int)time(NULL));
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

