#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

#define OBJECT_COUNT 4
#define BLOCK_DIM 1024
// includes, system
// #include <GL/glew.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <thrust/device_vector.h>
#include <bits/stdc++.h>


#define GLM_FORCE_CUDA
#include "glm/glm.hpp"


#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width    = 256;
const unsigned int mesh_height   = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";

struct object{
    int n_vertices;
    float4 speed;
}objects[OBJECT_COUNT];



std::vector<glm::vec3> vertices;
std::vector<glm::vec2> uvs;
std::vector<glm::vec3> normals;

bool loadOBJ(
    const char * path, 
    std::vector<glm::vec3> & out_vertices, 
    std::vector<glm::vec2> & out_uvs,
    std::vector<glm::vec3> & out_normals
);

__device__ int getObjectId(int index, struct object* d_objects){
    int sum = 0;
    for (int i = 0; i < OBJECT_COUNT; ++i)
    {
        sum = sum + d_objects[i].n_vertices;
        if(index < sum)
        {
            // if(i==0)
            // printf("Sent %d\n", i);
            return i;
        }

    }
    // printf("getObjectId: Object Id not found. %d %d %d %d Sending -1\n",index, sum, d_objects[0].n_vertices, d_objects[1].n_vertices);
    return -1;
}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, struct object* d_objects, float time)
{
    // unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    // unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    // printf("here\n");

    int object_id = getObjectId(idx, d_objects);
    if(object_id == -1)
        return;
    // printf("now here\n");
    
    float4 speed = d_objects[object_id].speed;
    // printf("%d: %f %f %f\n", object_id,speed.x, speed.y, speed.z);
    // printf("%d: %f %f %f\n", idx, pos[idx].x, pos[idx].y, pos[idx].z);
    pos[idx] = make_float4(pos[idx].x + speed.x*time, pos[idx].z + speed.z*time, 
        pos[idx].y+speed.y*time, 1.0f);
    // printf("%d: %f %f %f\n", idx, pos[idx].x, pos[idx].y, pos[idx].z);

    // pos[y*width+x] = make_float4(u, w, v, 1.0f);
}



void launch_kernel(float4 *pos, struct object* objects, float time)
{
    // execute the kernel
    dim3 grid(ceil((float)vertices.size()/BLOCK_DIM),1);
    dim3 block(BLOCK_DIM,1);
    simple_vbo_kernel<<< grid, block>>>(pos, objects, time);
}

bool checkHW(char *name, const char *gpuType, int dev)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    strcpy(name, deviceProp.name);

    if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
    {
        return true;
    }
    else
    {
        return false;
    }
}

int findGraphicsGPU(char *name)
{
    int nGraphicsGPU = 0;
    int deviceCount = 0;
    bool bFoundGraphics = false;
    char firstGraphicsName[256], temp[256];

    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("> FAILED %s sample finished, exiting...\n", sSDKsample);
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("> There are no device(s) supporting CUDA\n");
        return false;
    }
    else
    {
        printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
        printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

        if (bGraphics)
        {
            if (!bFoundGraphics)
            {
                strcpy(firstGraphicsName, temp);
            }

            nGraphicsGPU++;
        }
    }

    if (nGraphicsGPU)
    {
        strcpy(name, firstGraphicsName);
    }
    else
    {
        strcpy(name, "this hardware");
    }

    return nGraphicsGPU;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }


    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // command line mode only
    if (ref_file != NULL)
    {
        // This will pick the best possible CUDA capable device
        int devID = findCudaDevice(argc, (const char **)argv);

        // create VBO
        checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, mesh_width*mesh_height*4*sizeof(float)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;
    }
    else
    {
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        if (checkCmdLineFlag(argc, (const char **)argv, "device"))
        {
            if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
            {
                return false;
            }
        }
        else
        {
            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }

        // register callbacks
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

        // create VBO
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // run the cuda part
        runCuda(&cuda_vbo_resource);

        // start rendering mainloop
        glutMainLoop();
    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    // printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);
    
    // printf("vertices size: %d\n",vertices.size());

    float4 host_pos[vertices.size()];
    for (int i = 0; i<vertices.size(); ++i)
    {
        // printf("i = %d\n",i);
        // printf("vertices[i].x = %f\n",vertices[i].x);
        // printf("pos[i].x = %f\n", host_pos[i].x);
        host_pos[i] = make_float4(vertices[i].x,vertices[i].y, vertices[i].z,1.0f);
    }
    hipMemcpy(dptr, host_pos, sizeof(host_pos), hipMemcpyHostToDevice);
    struct object* d_objects;
    hipMalloc(&d_objects, OBJECT_COUNT * sizeof(object));
    hipMemcpy(d_objects, objects, sizeof(objects), hipMemcpyHostToDevice);
    launch_kernel(dptr, d_objects, g_fAnim);
    hipFree(d_objects);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(mesh_width*mesh_height*sizeof(float));

    // execute the kernel
    launch_kernel((float4 *)d_vbo_buffer, objects, g_fAnim);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, mesh_width*mesh_height*sizeof(float), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, mesh_width*mesh_height*sizeof(float), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                mesh_width*mesh_height*sizeof(float),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    // unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);


    bool res = loadOBJ("cone.obj", vertices, uvs, normals);
    if(res == false)
        exit(1);
    objects[0].n_vertices = vertices.size();
    objects[0].speed = make_float4(0.1f,0.0f,0.0f,1.0f);

    std::vector<glm::vec3> temp_vertices;
    res = loadOBJ("cube.obj", temp_vertices, uvs, normals);
    if(res == false)
        exit(1);
    objects[1].n_vertices = temp_vertices.size();
    objects[1].speed = make_float4(0.0f,0.0f,0.1f,1.0f);

    vertices.insert(vertices.end(), temp_vertices.begin(), temp_vertices.end());

    res = loadOBJ("cube.obj", temp_vertices, uvs, normals);
    if(res == false)
        exit(1);
    objects[2].n_vertices = temp_vertices.size();
    objects[2].speed = make_float4(0.0f,0.0f,0.2f,1.0f);

    vertices.insert(vertices.end(), temp_vertices.begin(), temp_vertices.end());

    res = loadOBJ("cone.obj", temp_vertices, uvs, normals);
    if(res == false)
        exit(1);
    objects[3].n_vertices = temp_vertices.size();
    objects[3].speed = make_float4(0.0f,0.1f,0.0f,1.0f);

    vertices.insert(vertices.end(), temp_vertices.begin(), temp_vertices.end());


    printf("Size of vertices %d\n",vertices.size());
    glBufferData(GL_ARRAY_BUFFER, vertices.size() * sizeof(float4), 0, GL_DYNAMIC_DRAW);
    // glBufferData(GL_ARRAY_BUFFER, vertices.size() * sizeof(float4), &vertices[0], GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_TRIANGLES, 0, vertices.size());
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        float *data = (float *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<float>("./data/regression.dat",
                                data, mesh_width * mesh_height * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}


bool loadOBJ(
    const char * path, 
    std::vector<glm::vec3> & out_vertices, 
    std::vector<glm::vec2> & out_uvs,
    std::vector<glm::vec3> & out_normals
){
    printf("Loading OBJ file %s...\n", path);

    std::vector<unsigned int> vertexIndices, uvIndices, normalIndices;
    std::vector<glm::vec3> temp_vertices; 
    std::vector<glm::vec2> temp_uvs;
    std::vector<glm::vec3> temp_normals;


    FILE * file = fopen(path, "r");
    if( file == NULL ){
        printf("Impossible to open the file ! Are you in the right path ? See Tutorial 1 for details\n");
        getchar();
        return false;
    }

    while( 1 ){

        char lineHeader[128];
        // read the first word of the line
        int res = fscanf(file, "%s", lineHeader);
        if (res == EOF)
            break; // EOF = End Of File. Quit the loop.

        // else : parse lineHeader
        
        if ( strcmp( lineHeader, "v" ) == 0 ){
            glm::vec3 vertex;
            fscanf(file, "%f %f %f\n", &vertex.x, &vertex.y, &vertex.z );
            temp_vertices.push_back(vertex);
        }
        // else if ( strcmp( lineHeader, "vt" ) == 0 ){
        //     glm::vec2 uv;
        //     fscanf(file, "%f %f\n", &uv.x, &uv.y );
        //     uv.y = -uv.y; // Invert V coordinate since we will only use DDS texture, which are inverted. Remove if you want to use TGA or BMP loaders.
        //     temp_uvs.push_back(uv);
        // }
        // else if ( strcmp( lineHeader, "vn" ) == 0 ){
        //     glm::vec3 normal;
        //     fscanf(file, "%f %f %f\n", &normal.x, &normal.y, &normal.z );
        //     temp_normals.push_back(normal);
        // }
        else if ( strcmp( lineHeader, "f" ) == 0 ){
            std::string vertex1, vertex2, vertex3;
            unsigned int vertexIndex[3], uvIndex[3], normalIndex[3];
            // int matches = fscanf(file, "%d/%d/%d %d/%d/%d %d/%d/%d\n", &vertexIndex[0], &uvIndex[0], &normalIndex[0], &vertexIndex[1], &uvIndex[1], &normalIndex[1], &vertexIndex[2], &uvIndex[2], &normalIndex[2] );
            int matches = fscanf(file, "%d %d %d\n", &vertexIndex[0], &vertexIndex[1], &vertexIndex[2]);
            // if (matches != 9){
            if (matches != 3){
                printf("File can't be read by our simple parser :-( Try exporting with other options\n");
                return false;
            }
            vertexIndices.push_back(vertexIndex[0]);
            vertexIndices.push_back(vertexIndex[1]);
            vertexIndices.push_back(vertexIndex[2]);
            // uvIndices    .push_back(uvIndex[0]);
            // uvIndices    .push_back(uvIndex[1]);
            // uvIndices    .push_back(uvIndex[2]);
            // normalIndices.push_back(normalIndex[0]);
            // normalIndices.push_back(normalIndex[1]);
            // normalIndices.push_back(normalIndex[2]);
        }else{
            // Probably a comment, eat up the rest of the line
            char stupidBuffer[1000];
            fgets(stupidBuffer, 1000, file);
        }

    }

    // For each vertex of each triangle
    for( unsigned int i=0; i<vertexIndices.size(); i++ ){

        // Get the indices of its attributes
        unsigned int vertexIndex = vertexIndices[i];
        // unsigned int uvIndex = uvIndices[i];
        // unsigned int normalIndex = normalIndices[i];
        
        // Get the attributes thanks to the index
        glm::vec3 vertex = temp_vertices[ vertexIndex-1 ];
        // glm::vec2 uv = temp_uvs[ uvIndex-1 ];
        // glm::vec3 normal = temp_normals[ normalIndex-1 ];
        
        // Put the attributes in buffers
        out_vertices.push_back(vertex);
        // out_uvs     .push_back(uv);
        // out_normals .push_back(normal);
    
    }

    return true;
}

