#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctime>
#include <climits>
#include <hip/hip_runtime_api.h>

#define R 16
#define L 8 //Don't increase L beyond 8
#define MAX_GRID_SIZE 2147483646
#define BLOCK_DIM_SORT 192
#define NUMBER_OF_GROUPS_PER_BLOCK 12
#define NUM_RADICES (1<<L)
#define NUM_BLOCKS 16//ceil((float)size/BLOCK_DIM_SORT) //If the size is 3072 = 192*16
#define ARRAY_SIZE 3072
#define NUM_GROUPS NUMBER_OF_GROUPS_PER_BLOCK * NUM_BLOCKS
#define NUM_RADICES_PER_BLOCK 16 // NUM_RADICES/NUM_BLOCK =  256/8 = 32

// int cellID[ARRAY_SIZE];
// int objectID[ARRAY_SIZE];



__device__ __host__ int getAddress(int a, int b, int c){
    //d1: radices
    //d2: thread blocks
    //d3: thread groups
    //a: radix, b: thread block, c: thread group

    // int d1 = NUM_RADICES;
	int d2 = NUM_BLOCKS;
	int d3 = NUMBER_OF_GROUPS_PER_BLOCK;
	return (d2*d3*a + d3*b + c);
    //http://stackoverflow.com/questions/789913/array-offset-calculations-in-multi-dimensional-array-column-vs-row-major
}

void __global__ phase_1_kernel(int *d_cellID, int *d_objectID, int size, int partition_size, int Num_Elements_Per_Group, int pass, int * d_counters){
	__shared__ int shared_counters[NUMBER_OF_GROUPS_PER_BLOCK][NUM_RADICES];
	if(threadIdx.x%R==0){
		for (int i = 0; i < NUM_RADICES; ++i){
			shared_counters[threadIdx.x/R][i] = 0;
		}
	}
	__syncthreads();
	int firstCellID =(blockIdx.x*NUMBER_OF_GROUPS_PER_BLOCK + threadIdx.x/R)*Num_Elements_Per_Group + threadIdx.x%R;
	unsigned int mask = 0;
	for (int i = 0; i < L; ++i)
		mask = mask<<1 | 1;
	mask = mask << (pass*L);
	for (int i = firstCellID; i < firstCellID + R*partition_size; i+=R)
	{
		unsigned int masked_number = d_cellID[i] & (mask);
		masked_number = masked_number >> (L*pass);
		atomicInc((unsigned int*)&shared_counters[threadIdx.x/R][masked_number], INT_MAX);
	}
	__syncthreads();
	if(threadIdx.x%R == 0){
		for (int i = 0; i < NUM_RADICES; ++i)
		{
			d_counters[getAddress(i,blockIdx.x,threadIdx.x/R)] = shared_counters[threadIdx.x/R][i];
		}
	}

}

void launch_kernel_phase_1(int* d_cellID, int * d_objectID, int size, int pass, int* d_counters){
    //keep launch config of all kernels same
	dim3 grid(ceil((float)size/BLOCK_DIM_SORT));
	dim3 block(BLOCK_DIM_SORT,1);
	int partition_size = ceil((float)size/MAX_GRID_SIZE);
	int Num_Elements_Per_Group = R*partition_size;
	phase_1_kernel <<<grid, block>>>(d_cellID, d_objectID, size, partition_size, Num_Elements_Per_Group, pass, d_counters);
}

void __global__ phase_2_kernel(int *d_cellID, int *d_objectID, int size, int partition_size, int Num_Elements_Per_Group, int pass, int * d_counters, int *d_partial_prefix_sums_per_radix){

	int lowestRadixForBlock = NUM_RADICES_PER_BLOCK*blockIdx.x ;
	int highestRadixForBlock = lowestRadixForBlock + NUM_RADICES_PER_BLOCK - 1;
    //Both radices are included in the radix range for this group
	__shared__ int shared_counters[NUM_RADICES_PER_BLOCK][NUM_GROUPS];
	if(threadIdx.x < NUM_RADICES_PER_BLOCK)
	{
		int i = threadIdx.x;
		for(int j = 0; j < NUM_GROUPS; j++)
		{
			shared_counters[i][j] = 0;
		}
	}
	__syncthreads();

	if(threadIdx.x <= highestRadixForBlock - lowestRadixForBlock)
	{
		int i = threadIdx.x + lowestRadixForBlock;
		for (int j = 0; j < NUM_GROUPS; ++j)
		{
			shared_counters[i - lowestRadixForBlock][j] = d_counters[getAddress(i,j/NUMBER_OF_GROUPS_PER_BLOCK, j%NUMBER_OF_GROUPS_PER_BLOCK)];
		}	
	}
	__syncthreads();

    //Prefix sum naive implementation for shared memory
	if(threadIdx.x <= highestRadixForBlock - lowestRadixForBlock)
	{
		int i = threadIdx.x + lowestRadixForBlock;
		for(int j = 1;j<NUM_GROUPS; j++){
			shared_counters[i - lowestRadixForBlock][j] = shared_counters[i - lowestRadixForBlock][j] + shared_counters[i - lowestRadixForBlock][j-1];
		}
	}
	__syncthreads();

	if(threadIdx.x <= highestRadixForBlock - lowestRadixForBlock)
	{
		int i = threadIdx.x + lowestRadixForBlock;
		for (int j = 0; j < NUM_GROUPS; ++j)
		{
			d_counters[getAddress(i,j/NUMBER_OF_GROUPS_PER_BLOCK, j%NUMBER_OF_GROUPS_PER_BLOCK)] = shared_counters[i - lowestRadixForBlock][j];
		}
		d_partial_prefix_sums_per_radix[i] = shared_counters[i - lowestRadixForBlock][NUM_GROUPS-1];
	}


}

void launch_kernel_phase_2(int* d_cellID, int * d_objectID, int size, int pass, int* d_counters, int* d_partial_prefix_sums_per_radix){
    //keep launch config of all kernels same
	dim3 grid(NUM_RADICES/NUM_RADICES_PER_BLOCK);
	dim3 block(BLOCK_DIM_SORT,1);
	int partition_size = ceil((float)size/MAX_GRID_SIZE);
	int Num_Elements_Per_Group = R*partition_size;
	phase_2_kernel <<<grid, block>>>(d_cellID, d_objectID, size, partition_size, Num_Elements_Per_Group, pass, d_counters, d_partial_prefix_sums_per_radix);
}


void __global__ phase_3_kernel(int *d_cellID, int *d_objectID, int size, int partition_size, int Num_Elements_Per_Group, int pass, int * d_counters, int* d_partial_prefix_sums_per_radix, int * d_sorted_cellID, int* d_sorted_objectID){

	__shared__ int shared_parallel_prefix[NUM_RADICES];
	__shared__ int shared_counters[NUMBER_OF_GROUPS_PER_BLOCK][NUM_RADICES];


	if(threadIdx.x%R==0){
		for (int i = 0; i < NUM_RADICES; ++i){
			shared_counters[threadIdx.x/R][i] = 0;
		}
	}
	__syncthreads();

	if(threadIdx.x == 0)
	{
		for (int i = 0; i < NUM_RADICES; ++i)
		{
			shared_parallel_prefix[i] = d_partial_prefix_sums_per_radix[i];
		}

	}
	__syncthreads();

    if(threadIdx.x == 0){
		for (int i = 1; i < NUM_RADICES; ++i)
		{
			shared_parallel_prefix[i] = shared_parallel_prefix[i-1] + shared_parallel_prefix[i];
		}
	}
	__syncthreads();

	if(threadIdx.x%R == 0){
		for (int i = 0; i < NUM_RADICES; ++i)
		{
			shared_counters[threadIdx.x/R][i] = d_counters[getAddress(i,blockIdx.x,threadIdx.x/R)];
			if(i>0)
				shared_counters[threadIdx.x/R][i] += shared_parallel_prefix[i-1];
		}
	}
	__syncthreads();

	int firstCellID =(blockIdx.x*NUMBER_OF_GROUPS_PER_BLOCK + threadIdx.x/R)*Num_Elements_Per_Group + threadIdx.x%R;
	unsigned int mask = 0;
	for (int i = 0; i < L; ++i)
		mask = mask<<1 | 1;
	mask = mask << (pass*L);
	for (int i = firstCellID; i < firstCellID + R*partition_size; i+=R)
	{
		int masked_number = d_cellID[i] & (mask);
		masked_number = masked_number >> (L*pass);
		if(shared_counters[threadIdx.x/R][masked_number] != ARRAY_SIZE) //TODO: Remove this if condition by fixing the math
			d_sorted_cellID[shared_counters[threadIdx.x/R][masked_number]] += masked_number << L*pass;
		masked_number = d_objectID[i] & (mask);
		masked_number = masked_number >> (L*pass);
		if(shared_counters[threadIdx.x/R][masked_number] != ARRAY_SIZE) //TODO: Remove this if condition by fixing the math
			d_sorted_objectID[shared_counters[threadIdx.x/R][masked_number]] += masked_number << L*pass; 
		atomicInc((unsigned int*)&shared_counters[threadIdx.x/R][masked_number], INT_MAX);
	}

}

void launch_kernel_phase_3(int* d_cellID, int * d_objectID, int size, int pass, int* d_counters, int *d_partial_prefix_sums_per_radix, int * d_sorted_cellID, int* d_sorted_objectID){
    //keep launch config of all kernels same
	dim3 grid(ceil((float)size/BLOCK_DIM_SORT));
	dim3 block(BLOCK_DIM_SORT,1);
	int partition_size = ceil((float)size/MAX_GRID_SIZE);
	int Num_Elements_Per_Group = R*partition_size;
	phase_3_kernel <<<grid, block>>>(d_cellID, d_objectID, size, partition_size, Num_Elements_Per_Group, pass, d_counters, d_partial_prefix_sums_per_radix,d_sorted_cellID, d_sorted_objectID);
}

void sort(int *d_cellID, int *d_objectID){

	// int * d_cellID, *d_objectID;
	// checkCudaErrors(hipMalloc(&d_cellID, ARRAY_SIZE*sizeof(int)));
	// checkCudaErrors(hipMalloc(&d_objectID, ARRAY_SIZE*sizeof(int)));
	// checkCudaErrors(hipMemcpy(d_cellID, cellID, ARRAY_SIZE*sizeof(int),hipMemcpyHostToDevice));
	// checkCudaErrors(hipMemcpy(d_objectID, objectID, ARRAY_SIZE*sizeof(int), hipMemcpyHostToDevice));

	int * d_counters;
	checkCudaErrors(hipMalloc(&d_counters, NUM_RADICES * NUM_BLOCKS * NUMBER_OF_GROUPS_PER_BLOCK * sizeof(int)));
	int* d_partial_prefix_sums_per_radix;
	checkCudaErrors(hipMalloc(&d_partial_prefix_sums_per_radix, sizeof(int) * NUM_RADICES));
	int *d_sorted_cellID;
	checkCudaErrors(hipMalloc(&d_sorted_cellID, ARRAY_SIZE*sizeof(int)));
	checkCudaErrors(hipMemset(d_sorted_cellID, 0, ARRAY_SIZE*sizeof(int)));
	int *d_sorted_objectID;
	checkCudaErrors(hipMalloc(&d_sorted_objectID, ARRAY_SIZE*sizeof(int)));
	checkCudaErrors(hipMemset(d_sorted_objectID, 0, ARRAY_SIZE*sizeof(int)));
	for(int i = 0; i < 4 ; i++)
	{	
		//printf("Pass %d\n", i);
		checkCudaErrors(hipMemset(d_counters, 0, NUM_RADICES * NUM_BLOCKS * NUMBER_OF_GROUPS_PER_BLOCK * sizeof(int)  ));
		launch_kernel_phase_1(d_cellID, d_objectID, ARRAY_SIZE, i, d_counters);

		// int *h_d_counters;
		// h_d_counters = (int *) malloc(NUM_RADICES * NUM_BLOCKS * NUMBER_OF_GROUPS_PER_BLOCK * sizeof(int));
		// checkCudaErrors(hipMemcpy(h_d_counters, d_counters, NUM_RADICES * NUM_BLOCKS * NUMBER_OF_GROUPS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost ));
	    // for (int i = 0; i < NUM_RADICES; ++i)
	    // {
	    // 	printf("Radix: %d Values: ", i);
	    //     for(int j = 0; j<NUM_BLOCKS; j++){
	    //         for(int k = 0; k<NUMBER_OF_GROUPS_PER_BLOCK; k++){
	    //             printf("%d ", h_d_counters[getAddress(i,j,k)]);
	    //         }
	    //         printf("\t");
	    //     }
	    //     printf("\n\n");
	    // }

		launch_kernel_phase_2(d_cellID, d_objectID, ARRAY_SIZE, i, d_counters, d_partial_prefix_sums_per_radix);

		int *h_d_partial_prefix_sums_per_radix;
		h_d_partial_prefix_sums_per_radix = (int*) malloc(sizeof(int) * NUM_RADICES);
		checkCudaErrors(hipMemcpy(h_d_partial_prefix_sums_per_radix, d_partial_prefix_sums_per_radix, sizeof(int) * NUM_RADICES, hipMemcpyDeviceToHost));
		//for (int i = 0; i < NUM_RADICES; ++i)
		//	printf("Radix %d: %d\n", i,h_d_partial_prefix_sums_per_radix[i]);

		launch_kernel_phase_3(d_cellID, d_objectID, ARRAY_SIZE, i, d_counters, d_partial_prefix_sums_per_radix, d_sorted_cellID, d_sorted_objectID);
	
	}

	int* h_d_sorted_cellID;
	h_d_sorted_cellID = (int *)malloc( ARRAY_SIZE* sizeof(int));
	checkCudaErrors(hipMemcpy(h_d_sorted_cellID, d_sorted_cellID, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost));
	// printf("Sorted Array\n");
	// for (int i = 0; i < ARRAY_SIZE; ++i)
	// {
	// 	printf("%d ", h_d_sorted_cellID[i]);
	// }

	int* h_d_sorted_objectID;
	h_d_sorted_objectID = (int *)malloc( ARRAY_SIZE* sizeof(int));
	checkCudaErrors(hipMemcpy(h_d_sorted_objectID, d_sorted_objectID, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost));

	// printf("\n");
	//printf("Sorted Array\n");
	//for (int i = 0; i < ARRAY_SIZE; ++i)
	//{
	//	printf("(%d, %d), ",h_d_sorted_cellID[i], h_d_sorted_objectID[i]);
	//}

	checkCudaErrors(hipFree(d_sorted_cellID));
	checkCudaErrors(hipFree(d_sorted_objectID));
	checkCudaErrors(hipFree(d_counters));

	


}

// int main(int argc, char const *argv[])
// {
// 	hipSetDevice(1);
// 	// srand(time(NULL));
// 	// for (int i = 0; i < ARRAY_SIZE; ++i)
// 	// {
// 	// 	cellID[i] = rand();
// 	// 	objectID[i] = i;
// 	// }
// 	// for (int i = 0; i < ARRAY_SIZE; ++i)
// 	// {
// 	// 	cellID[i] = i;
// 	// 	objectID[i] = ARRAY_SIZE - i;
// 	// }
// 	sort();
// 	return 0;
// }
